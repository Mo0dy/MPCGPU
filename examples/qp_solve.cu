#include "hip/hip_runtime.h"
#include <fstream>      // std::ofstream
#include <iostream>
#include <stdio.h>
#include "gpuassert.cuh"
#include "read_array.h"
#include <ctime>
#include "pcg/qp.cuh"
#include <tuple>

#define tic      double tic_t = clock();
#define toc      std::cout << (clock() - tic_t)/CLOCKS_PER_SEC \
                           << " seconds" << std::endl;

int main() {

    const uint32_t state_size = STATE_SIZE;
    const uint32_t knot_points = KNOT_POINTS;
    const uint32_t control_size = CONTROL_SIZE;

    const uint32_t states_sq = state_size * state_size;
    const uint32_t states_p_controls = state_size * control_size;
    const uint32_t controls_sq = control_size * control_size;
    const uint32_t states_s_controls = state_size + control_size;

    const uint32_t KKT_G_DENSE_SIZE = (states_sq + controls_sq) * knot_points - controls_sq;
    const uint32_t KKT_C_DENSE_SIZE = (states_sq + states_p_controls) * (knot_points - 1);
    const uint32_t KKT_g_SIZE = (state_size + control_size) * knot_points - control_size;
    const uint32_t KKT_c_SIZE = state_size * knot_points;
    const uint32_t DZ_SIZE = states_s_controls * knot_points - control_size;

    double h_G_dense[KKT_G_DENSE_SIZE];
    double h_C_dense[KKT_C_DENSE_SIZE];
    double h_g[KKT_g_SIZE];
    double h_c[KKT_c_SIZE];
    readArrayFromFile(KKT_G_DENSE_SIZE, "data/G_dense.txt", h_G_dense);
    readArrayFromFile(KKT_C_DENSE_SIZE, "data/C_dense.txt", h_C_dense);
    readArrayFromFile(KKT_g_SIZE, "data/g.txt", h_g);
    readArrayFromFile(KKT_c_SIZE, "data/c.txt", h_c);

    double h_dz_trans[DZ_SIZE];
    double h_dz_org[DZ_SIZE];

    struct pcg_config<double> config;
    config.pcg_org_trans = false;
    std::tuple<uint32_t, double, double> qp_trans_stats_m0, qp_trans_stats_m1, qp_org_stats_m0, qp_org_stats_m1;
    config.pcg_poly_order = 0;
    qp_org_stats_m0 = qpSolvePcg<double>(state_size, control_size, knot_points,
                                         h_G_dense,
                                         h_C_dense,
                                         h_g,
                                         h_c,
                                         h_dz_org,
                                         CHOL_OR_LDL,
                                         config);

    config.pcg_poly_order = 1;
    config.pcg_poly_coeff[0] = 1.0;
    qp_org_stats_m1 = qpSolvePcg<double>(state_size, control_size, knot_points,
                                         h_G_dense,
                                         h_C_dense,
                                         h_g,
                                         h_c,
                                         h_dz_org,
                                         CHOL_OR_LDL,
                                         config);

    config.pcg_org_trans = true;
    config.pcg_poly_order = 0;
    qp_trans_stats_m0 = qpSolvePcg<double>(state_size, control_size, knot_points,
                                           h_G_dense,
                                           h_C_dense,
                                           h_g,
                                           h_c,
                                           h_dz_trans,
                                           CHOL_OR_LDL,
                                           config);

    config.pcg_poly_order = 1;
    config.pcg_poly_coeff[0] = 1.0;
    qp_trans_stats_m1 = qpSolvePcg<double>(state_size, control_size, knot_points,
                                           h_G_dense,
                                           h_C_dense,
                                           h_g,
                                           h_c,
                                           h_dz_trans,
                                           CHOL_OR_LDL,
                                           config);

    uint32_t pcg_org_iters_m0 = std::get<0>(qp_org_stats_m0);
    uint32_t pcg_org_iters_m1 = std::get<0>(qp_org_stats_m1);
    uint32_t pcg_trans_iters_m0 = std::get<0>(qp_trans_stats_m0);
    uint32_t pcg_trans_iters_m1 = std::get<0>(qp_trans_stats_m1);

    std::cout << "Original PCG iteration number m = 0: " << pcg_org_iters_m0 << std::endl;
    std::cout << "Original PCG iteration number m = 1: " << pcg_org_iters_m1 << std::endl;
    std::cout << "Transformed PCG iteration number m = 0: " << pcg_trans_iters_m0 << std::endl;
    std::cout << "Transformed PCG iteration number m = 1: " << pcg_trans_iters_m1 << std::endl;

    double norm_org = 0;
    double norm_trans = 0;
    double diff = 0;
    for (uint32_t i = 0; i < DZ_SIZE; i++) {
        norm_org += h_dz_org[i] * h_dz_org[i];
        norm_trans += h_dz_trans[i] * h_dz_trans[i];
        diff += (h_dz_org[i] - h_dz_trans[i]) * (h_dz_org[i] - h_dz_trans[i]);
    }
    std::cout << "Original dz norm: " << sqrt(norm_org) << std::endl;
    std::cout << "Transformed dz norm: " << sqrt(norm_trans) << std::endl;
    std::cout << "dz norm difference: " << sqrt(diff) << std::endl;


//    int iteration = 1000;
//    double linsys_time_total = 0;
//    double qp_solve_time_total = 0;
//    for (int i = 0; i < iteration; i++) {
//        qp_trans_stats = qpSolvePcg<double, CHOL_OR_LDL>(state_size, control_size, knot_points,
//                                                             h_G_dense,
//                                                             h_C_dense,
//                                                             h_g,
//                                                             h_c,
//                                                             h_dz_trans,
//                                                             config);
//        double linsys_time = std::get<1>(qp_trans_stats);
//        double qp_solve_time = std::get<2>(qp_trans_stats);
//        linsys_time_total += linsys_time;
//        qp_solve_time_total += qp_solve_time;
//    }
//    std::cout << "PCG time avg in " << iteration << " iterations: " << linsys_time_total / iteration
//              << " us (1e-6) microseconds. " << std::endl;
//    std::cout << "QP time avg in " << iteration << " iterations: " << qp_solve_time_total / iteration
//              << " us (1e-6) microseconds. " << std::endl;
    return 0;
}

