#include "hip/hip_runtime.h"
#include <fstream>      // std::ofstream
#include <iostream>
#include <stdio.h>
#include "gpuassert.cuh"
#include "read_array.h"
#include <ctime>
#include "pcg/qp.cuh"
#include <tuple>

#define tic      double tic_t = clock();
#define toc      std::cout << (clock() - tic_t)/CLOCKS_PER_SEC \
                           << " seconds" << std::endl;

int main() {

    const uint32_t state_size = STATE_SIZE;
    const uint32_t knot_points = KNOT_POINTS;
    const uint32_t control_size = CONTROL_SIZE;

    const uint32_t states_sq = state_size * state_size;
    const uint32_t states_p_controls = state_size * control_size;
    const uint32_t controls_sq = control_size * control_size;
    const uint32_t states_s_controls = state_size + control_size;

    const uint32_t KKT_G_DENSE_SIZE = (states_sq + controls_sq) * knot_points - controls_sq;
    const uint32_t KKT_C_DENSE_SIZE = (states_sq + states_p_controls) * (knot_points - 1);
    const uint32_t KKT_g_SIZE = (state_size + control_size) * knot_points - control_size;
    const uint32_t KKT_c_SIZE = state_size * knot_points;
    const uint32_t DZ_SIZE = states_s_controls * knot_points - control_size;

    double h_G_dense[KKT_G_DENSE_SIZE];
    double h_C_dense[KKT_C_DENSE_SIZE];
    double h_g[KKT_g_SIZE];
    double h_c[KKT_c_SIZE];
    readArrayFromFile(KKT_G_DENSE_SIZE, "data/G_dense.txt", h_G_dense);
    readArrayFromFile(KKT_C_DENSE_SIZE, "data/C_dense.txt", h_C_dense);
    readArrayFromFile(KKT_g_SIZE, "data/g.txt", h_g);
    readArrayFromFile(KKT_c_SIZE, "data/c.txt", h_c);

    double h_dz[DZ_SIZE];
    // set zeros to dz. Not necessary.
    for (uint32_t i = 0; i < DZ_SIZE; i++) {
        h_dz[i] = 0;
    }

    struct pcg_config<double> config;
    config.pcg_org_trans = false;
    config.pcg_poly_order = 0;
    std::tuple<uint32_t, double, double> qp_stats;
    qp_stats = qpSolvePcg<double>(state_size, control_size, knot_points,
                                 h_G_dense,
                                 h_C_dense,
                                 h_g,
                                 h_c,
                                 h_dz,
                                 config);
    uint32_t pcg_iters = std::get<0>(qp_stats);

    std::cout << "PCG iteration number: " << pcg_iters << std::endl;

    double norm = 0;
    for (uint32_t i = 0; i < DZ_SIZE; i++) {
        norm += h_dz[i] * h_dz[i];
    }
    std::cout << "dz norm: " << sqrt(norm) << std::endl;

//    int iteration = 1000;
//    double linsys_time_total = 0;
//    double qp_solve_time_total = 0;
//    for (int i = 0; i < iteration; i++) {
//        qp_stats = qpSolvePcg(state_size, control_size, knot_points,
//                              h_G_dense,
//                              h_C_dense,
//                              h_g,
//                              h_c,
//                              h_dz,
//                              config);
//        double linsys_time = std::get<1>(qp_stats);
//        double qp_solve_time = std::get<2>(qp_stats);
//        linsys_time_total += linsys_time;
//        qp_solve_time_total += qp_solve_time;
//    }
//    std::cout << "PCG time avg in " << iteration << " iterations: " << linsys_time_total / iteration
//              << " us (1e-6) microseconds. " << std::endl;
//    std::cout << "QP time avg in " << iteration << " iterations: " << qp_solve_time_total / iteration
//              << " us (1e-6) microseconds. " << std::endl;

    return 0;
}

